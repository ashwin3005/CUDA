
#include <hip/hip_runtime.h>
#include <stdio.h>

int matMul(float *A, float *B, float *RES, int arow, int acol, int brow, int bcol){
    if(acol != brow){
        printf("Not possible for multipling the matrices, check for the dimensions !");
        return 1;
    }
    int res_row = arow;
    int res_col = bcol;

    for(int i=0; i<res_row; i++){
        for(int j=0; j<res_col; j++){
            float dotprod = 0.0f;
            for(int k=0; k<acol; k++){
                dotprod += A[i*acol + k] * B[k*bcol + j];
            }
            RES[i*res_col + j] = dotprod;   
        }
    }
    return 0;
}
void printMatrix(float *A, int rows, int cols);
int main(){
    int arow = 2;
    int acol = 3;
    int brow = 3;
    int bcol = 2;
    float *A, *B, *RES;
    A = (float*)malloc(arow*acol*sizeof(float));
    B = (float*)malloc(brow*bcol*sizeof(float));
    RES = (float*)malloc(arow*bcol*sizeof(float));

    printf("Enter the inputs for A matrix.\n");
    for(int i=0; i<arow; i++){
        for(int j=0; j<acol; j++){
            float in;
            printf("Enter value for (%d, %d): ", i, j);
            scanf("%f", &in);
            A[i* acol + j] = in;
        }
    }
    printf("Enter the inputs for B matrix.\n");
    for(int i=0; i<brow; i++){
        for(int j=0; j<bcol; j++){
            float in;
            printf("Enter value for (%d, %d): ", i, j);
            scanf("%f", &in);
            B[i* bcol + j] = in;
        }
    }
    matMul(A, B, RES, arow, acol, brow, bcol);

    printf("Matrix A:\n");
    printMatrix(A, arow, acol);
    printf("Matrix B:\n");
    printMatrix(B, brow, bcol);
    printf("Result Matrix:\n");
    printMatrix(RES, arow, bcol);
}

void printMatrix(float *A, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%f ", A[i * cols + j]);
        }
        printf("\n");
    }
    printf("\n");
}