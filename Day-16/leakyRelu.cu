
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// Leaky ReLU kernel
__global__ void leakyReLU_kernel(float *data_d, float alpha, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        data_d[idx] = (data_d[idx] > 0) ? data_d[idx] : alpha * data_d[idx];
    }
}

int main() {
    int n = 16; 
    float alpha = 0.01f; // Leaky ReLU slope

    float *data_h = (float *)malloc(n * sizeof(float)); 
    float *data_d; 
    srand(time(NULL)); 

    for (int i = 0; i < n; i++) {
        data_h[i] = (float)rand()/float(RAND_MAX) * 100.0f; 
    }

    hipMalloc((void **)&data_d, n * sizeof(float));
    hipMemcpy(data_d, data_h, n * sizeof(float), hipMemcpyHostToDevice);


    leakyReLU_kernel<<<(n + 255) / 256, 256>>>(data_d, alpha, n);
    

    hipMemcpy(data_h, data_d, n * sizeof(float), hipMemcpyDeviceToHost);

    for(int i=0; i<n; i++){
        printf("%f ", data_d[i]);
    }

    // free 
    hipFree(data_d);
    free(data_h);

    return 0;
}
