#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "utils.h"

__global__ void squared_matmul(float *A, float *B, float *C, int width){
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    if (row<width && col < width){
        float dotprod = 0.0f;
        for(int i=0; i<width; i++){
            dotprod += A[row*width + i] * B[i*width + col];
        }C[row*width + col] =  dotprod;
    }
}

__global__ void unco_squared_matmul(float *A, float *B, float *C, int width){
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int col = blockDim.y * blockIdx.y + threadIdx.y;
    if (row<width && col < width){
        float dotprod = 0.0f;
        for(int i=0; i<width; i++){
            dotprod += A[row*width + i] * B[i*width + col];
        }C[row*width + col] =  dotprod;
    }
}

struct timer t;

int main(){

    int N = 10000; // 10k
    int bytes = N * N * sizeof(float);
    srand(time(NULL));

    float *A, *B, *C;

    A = (float*)malloc(bytes);
    B = (float*)malloc(bytes);
    C = (float*)malloc(bytes);

    for(int i=0; i<N*N; i++){
        A[i] = ((float)rand()/ (float)RAND_MAX) * 200.0f - 100.0f;
        B[i] = ((float)rand()/ (float)RAND_MAX) * 200.0f - 100.0f;
    }

    float *A_d, *B_d, *C_d;
    hipMalloc((void**)&A_d, bytes);
    hipMalloc((void**)&B_d, bytes);
    hipMalloc((void**)&C_d, bytes);

    hipMemcpy(A_d, A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, bytes, hipMemcpyHostToDevice);

    // launch kernel
    dim3 THREADS(32, 32, 1);
    dim3 BLOCKS((N + THREADS.x - 1/THREADS.x),(N + THREADS.x - 1/THREADS.x),1);

    start_timer(&t);
    squared_matmul<<<BLOCKS, THREADS>>>(A_d, B_d, C_d, N);
    hipDeviceSynchronize();
    stop_timer(&t);

    printf("Time taken for mat-mul in row major %f seconds\n", time_diff(&t));

    start_timer(&t);
    unco_squared_matmul<<<BLOCKS, THREADS>>>(A_d, B_d, C_d, N);
    hipDeviceSynchronize();
    stop_timer(&t);

    printf("Time taken for mat-mul in col major %f seconds\n", time_diff(&t));

    free(A);
    free(B);
    free(C);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    return 0;
}