#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void linear_layer_d(float* input_d, float* output_d, float* weights_d, float* bias_d, int input_size, int output_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < output_size) {
        output_d[i] = 0.0f;
        for (int j = 0; j < input_size; j++) {
            output_d[i] += input_d[j] * weights_d[i * input_size + j];
        }
        output_d[i] += bias_d[i];
    }
}

int main() {
    int input_size = 3;   
    int output_size = 2;  

    float input_h[3] = {1.0, 2.0, 3.0};

    float weights_h[6] = {0.1, 0.2, 0.3, 0.4, 0.5, 0.6};  

    float bias_h[2] = {0.1, 0.2};

    float output_h[2];

    float *input_d, *output_d, *weights_d, *bias_d;

    hipMalloc((void**)&input_d, input_size * sizeof(float));
    hipMalloc((void**)&output_d, output_size * sizeof(float));
    hipMalloc((void**)&weights_d, input_size * output_size * sizeof(float));
    hipMalloc((void**)&bias_d, output_size * sizeof(float));

    hipMemcpy(input_d, input_h, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(weights_d, weights_h, input_size * output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(bias_d, bias_h, output_size * sizeof(float), hipMemcpyHostToDevice);

    int block_size = 2;
    int grid_size = (output_size + block_size - 1) / block_size;

    // Launch the kernel
    linear_layer_d<<<grid_size, block_size>>>(input_d, output_d, weights_d, bias_d, input_size, output_size);

    // Copy the result from device to host
    hipMemcpy(output_h, output_d, output_size * sizeof(float), hipMemcpyDeviceToHost);

    printf("Output: ");
    for (int i = 0; i < output_size; i++) {
        printf("%f ", output_h[i]);
    }
    printf("\n");

    hipFree(input_d);
    hipFree(output_d);
    hipFree(weights_d);
    hipFree(bias_d);

    return 0;
}
