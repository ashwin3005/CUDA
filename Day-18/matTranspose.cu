#include <hip/hip_runtime.h>
#include <stdio.h>

#define WIDTH 1024
#define HEIGHT 1024

// CUDA kernel for matrix transposition
__global__ void transposeMatrix(const float* input, float* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        output[y * width + x] = input[x * height + y];
    }
}

int main() {
    int width = WIDTH;
    int height = HEIGHT;
    size_t size = width * height * sizeof(float);

    // Allocate host memory
    float* h_input = (float*)malloc(size);
    float* h_output = (float*)malloc(size);

    // Initialize the input matrix
    for (int i = 0; i < width * height; i++) {
        h_input[i] = (float)i;
    }

    // Allocate device memory
    float* d_input;
    float* d_output;
    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);

    // Copy input data from host to device
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    // Define block and grid size
    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Launch the kernel
    transposeMatrix<<<gridSize, blockSize>>>(d_input, d_output, width, height);

    // Wait for the kernel to finish
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);

    // Verify result
    int success = 1;
    for (int i = 0; i < width && success; i++) {
        for (int j = 0; j < height; j++) {
            if (h_output[i * height + j] != h_input[j * width + i]) {
                success = 0;
                break;
            }
        }
    }

    // Output result using printf
    if (success) {
        printf("Matrix transposition succeeded!\n");
    } else {
        printf("Matrix transposition failed!\n");
    }

    // Free memory
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);

    return 0;
}
